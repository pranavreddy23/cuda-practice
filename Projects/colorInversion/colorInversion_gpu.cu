#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "../common/profiler.h"
#include "colorInversion.h"
#include <iostream>
#include <float.h>

// Helper function for checking CUDA errors
#define CHECK_CUDA_ERROR(call) { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        std::cerr << "CUDA error in " << __FILE__ << " at line " << __LINE__ << ": " \
                  << hipGetErrorString(err) << std::endl; \
        exit(1); \
    } \
}

__global__ void invert_kernel(unsigned char* image, int width, int height) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if the index is within the bounds of the image
    if (idx < width * height) {
        // Each pixel has 4 components: R, G, B, A
        int pixelIndex = idx * 4;

        // Invert the R, G, B components (subtract from 255)
        image[pixelIndex] = 255 - image[pixelIndex];       // Invert Red
        image[pixelIndex + 1] = 255 - image[pixelIndex + 1]; // Invert Green
        image[pixelIndex + 2] = 255 - image[pixelIndex + 2]; // Invert Blue
        // Alpha remains unchanged, so we do not modify image[pixelIndex + 3]
    }
}

// image_input, image_output are device pointers (i.e. pointers to memory on the GPU)
void colorInversion_gpu(unsigned char* image, int width, int height) {
    static Profiler::MemoryTracker& mem_tracker = Profiler::MemoryTracker::getInstance();
    Profiler::GPUTimer total_timer;
    Profiler::GPUTimer kernel_timer;
    
    total_timer.start();
    unsigned char *d_image;
    CHECK_CUDA_ERROR(hipMalloc(&d_image, sizeof(unsigned char) * width * height * 4));
    
    mem_tracker.record_gpu_allocation(sizeof(unsigned char) * width * height * 4);
    
    CHECK_CUDA_ERROR(hipMemcpy(d_image, image, sizeof(unsigned char) * width * height * 4, hipMemcpyHostToDevice));
    
    int threadsPerBlock = 256;
    int blocksPerGrid = (width * height + threadsPerBlock - 1) / threadsPerBlock;

    kernel_timer.start();
    invert_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_image, width, height);
    CHECK_CUDA_ERROR(hipGetLastError()); // Check for launch errors
    CHECK_CUDA_ERROR(hipDeviceSynchronize());
    kernel_timer.stop();
    Profiler::KernelTimeTracker::record_kernel_time("colorInversion", kernel_timer.elapsed_milliseconds());

    CHECK_CUDA_ERROR(hipMemcpy(image, d_image, sizeof(unsigned char) * width * height * 4, hipMemcpyDeviceToHost));
    
    CHECK_CUDA_ERROR(hipFree(d_image));
    total_timer.stop();
    Profiler::KernelTimeTracker::last_total_time = total_timer.elapsed_milliseconds();
}